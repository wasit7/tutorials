//#include "device.h"
//#include <cutil.h>
//#include <cutil_math.h>
//#include <cutil_inline_runtime.h>

//#include <cutil.h>
//#include <cutil_math.h>
//#include <cutil_inline_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>

__global__ void g_draw(hipSurfaceObject_t sf,float time,unsigned int width,unsigned height){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    g_image[((y*width+x)*4+0)*sizeof(unsigned char)]= 125;//(int)(time*60)%255;
//    g_image[((y*width+x)*4+1)*sizeof(unsigned char)]= 255;//255*x/width;
//    g_image[((y*width+x)*4+2)*sizeof(unsigned char)]= 125;//255-255*x/width;
//    g_image[((y*width+x)*4+3)*sizeof(unsigned char)]= 105;//0;
    float4 data = make_float4(.0f, .6f, .8f, 1.0f);
    surf2Dwrite(data, sf, x * sizeof(float4), y);
}
extern "C" void mydraw(hipSurfaceObject_t sf, float time,unsigned int width, unsigned height){
    dim3 dimBlock(32,32,1);
	dim3 dimGrid(width/dimBlock.x,height/dimBlock.y,1);
    g_draw<<<dimGrid,dimBlock>>>(sf,time,width,height);
    //cutilSafeCall( cudaThreadSynchronize() );
}
__global__ void g_setKernel(hipSurfaceObject_t outputSurfObj,int width, int height){
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        data=make_uchar4(32,64,128,255);
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}
extern "C" void setKernel(hipSurfaceObject_t outputSurfObj,int width, int height){
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,(height + dimBlock.y - 1) / dimBlock.y);
    g_setKernel<<<dimGrid, dimBlock>>>(outputSurfObj, width, height);
}

