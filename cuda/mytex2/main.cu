#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <windows.h>
#include <GL/glew.h>
#include <GL/glut.h>
//#include "SOIL.H"
#include "device.h"

//#include <cutil_inline.h>
//#include <cutil_gl_inline.h>
extern "C" void mydraw (hipSurfaceObject_t sf, float time,unsigned int width, unsigned height);
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
unsigned int width=128;
unsigned int height=96;
GLuint gl_tex,gl_buffer;
struct hipGraphicsResource* cuda_tex;
hipError_t e;
void displayCB();
void mouseCB(int button, int state, int x, int y);
void mouseMotionCB(int x, int y);
void reshapeCB(int w, int h);
//view parameter
bool mouseLeftDown;
bool mouseRightDown;
float mouseX, mouseY;
float cameraAngleX;
float cameraAngleY;
float cameraDistance;

__global__ void setKernel(hipSurfaceObject_t outputSurfObj,
                           int width, int height)
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        data=make_uchar4(32,64,128,255);
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

int main(int argc, char **argv){
    // Initialize OpenGL and GLUT for device 0
    // and make the OpenGL context current
    printf("debug00:\n");
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowPosition(100,100);
    glutInitWindowSize(512,512);
    glutCreateWindow(argv[0]);
    glutDisplayFunc(displayCB);
    glutMouseFunc(mouseCB);
    glutMotionFunc(mouseMotionCB);
    glutReshapeFunc(reshapeCB);
    glEnable(GL_TEXTURE_2D);

    //gen gl_tex
    glGenTextures(1, &gl_tex);
    //bind gl_tex
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);

    //update gl_tex
    float pixels[] = {
        0.0f, 0.0f, 0.0f,   1.0f, 1.0f, 1.0f,
        1.0f, 1.0f, 1.0f,   0.0f, 0.0f, 0.0f
    };
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, 2, 2, 0, GL_RGB, GL_FLOAT, pixels);
    glBindTexture(GL_TEXTURE_2D, 0);


    hipSetDevice(0);\
    //hipGraphicsGLRegisterImage(&cuda_tex,gl_tex,GL_TEXTURE_2D,cudaGraphicsMapFlagsWriteDiscard)
//    glGenBuffersARB(1,&gl_buffer);
    //glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB,gl_buffer);
    //unsigned int size=width*height*4*sizeof(unsigned char);
    //glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB,size,0,GL_DYNAMIC_DRAW);
    //glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB,0);
    e = hipGraphicsGLRegisterImage(&cuda_tex, gl_tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    e = hipGraphicsGLRegisterBuffer(&cuda_tex,gl_buffer,cudaGraphicsMapFlagsWriteDiscard);
    //main loop
    glutMainLoop();
}
void displayCB(){
    //cuda write tex
//    unsigned char* g_image;

    hipGraphicsMapResources(1,&cuda_tex,0);
    hipArray_t cuda_array;
    e =  hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_tex, 0, 0);


    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj, &resDesc);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    setKernel<<<dimGrid, dimBlock>>>(outputSurfObj,width, height);

    //size_t num_bytes;

    //hipGraphicsResourceGetMappedPointer((void**)&g_image,&num_bytes,cuda_tex);

    //mydraw(g_image,glutGet(GLUT_ELAPSED_TIME)*1e-3f,width,height);
    hipGraphicsUnmapResources(1,&cuda_tex,0);


    //glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, gl_buffer);
//    glBindTexture(GL_TEXTURE_2D, gl_tex);
//    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
//    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);


    // tramsform camera
    glClearColor(0.5f,0.5f,0.5f,0.5f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0,0,1,0,0,0,0,1,0);
    glTranslatef(0, 0, cameraDistance);
    glRotatef(cameraAngleX, 1, 0, 0);   // pitch
    glRotatef(cameraAngleY, 0, 1, 0);   // heading




    //render
    /*
    float pixels[] = {
        0.0f, 0.0f, 0.0f,   1.0f, 1.0f, 1.0f,
        1.0f, 1.0f, 1.0f,   0.0f, 0.0f, 0.0f
    };
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, 2, 2, 0, GL_RGB, GL_FLOAT, pixels);
    */
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glBegin(GL_QUADS);
    glNormal3f(0, 0, 1);
    //glColor4f(0.5, 0.5, 0.5, 0.5);
    glTexCoord2f(0.0f, 0.0f);   glVertex3f(-1.0f, -1.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);   glVertex3f( 1.0f, -1.0f, 0.0f);
    glTexCoord2f(1.0f, 1.0f);   glVertex3f( 1.0f,  1.0f, 0.0f);
    glTexCoord2f(0.0f, 1.0f);   glVertex3f(-1.0f,  1.0f, 0.0f);
    glEnd();


    glBegin(GL_TRIANGLES);
    glNormal3f(0, 0, 1);
    glTexCoord2f(0.0f, 0.0f);   glVertex3f(-1.0f, -1.0f, 1.0f);
    glTexCoord2f(1.0f, 1.0f);   glVertex3f( 1.0f,  1.0f, 1.0f);
    glTexCoord2f(1.0f, 0.0f);   glVertex3f( 1.0f, -1.0f, 1.0f);
    glEnd();
    //glBindTexture(GL_TEXTURE_2D, 0);


    glutSwapBuffers();
    glutPostRedisplay();
}
void mouseCB(int button, int state, int x, int y){
    mouseX = (float)x;
    mouseY = (float)y;

    if(button == GLUT_LEFT_BUTTON)
    {
        if(state == GLUT_DOWN)
        {
            mouseLeftDown = true;
        }
        else if(state == GLUT_UP)
            mouseLeftDown = false;
    }

    else if(button == GLUT_RIGHT_BUTTON)
    {
        if(state == GLUT_DOWN)
        {
            mouseRightDown = true;
        }
        else if(state == GLUT_UP)
            mouseRightDown = false;
    }
}
void mouseMotionCB(int x, int y){
    if(mouseLeftDown)
    {
        cameraAngleY += 0.2f*(x - mouseX);
        cameraAngleX += 0.2f*(y - mouseY);
        mouseX = (float)x;
        mouseY = (float)y;
    }
    if(mouseRightDown)
    {
        cameraDistance += (y - mouseY) * 0.1f;
        mouseY = (float)y;
    }
    //printf("cameraDistance %f%\n",cameraDistance);
    glutPostRedisplay();
}
void reshapeCB(int w, int h){
    // set viewport to be the entire window
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);

    // set perspective viewing frustum
   glMatrixMode(GL_PROJECTION);
   glLoadIdentity();
    //glFrustum(-aspectRatio, aspectRatio, -1, 1, 1, 100);
   gluPerspective(36.0f, (float)(w)/h, 0.0f, 1000.0f); // FOV, AspectRatio, NearClip, FarClip

    // switch to modelview matrix in order to set scene
    glMatrixMode(GL_MODELVIEW);
}
