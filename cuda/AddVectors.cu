#include "hip/hip_runtime.h"
#include <iostream>
#define W 640 //width of the image
#define H 480 //height of the image
__global__ void kernelAdd(int A[H][W], int B[H][W], int C[H][W]){
	int x=blockIdx.x*blockDim.x + threadIdx.x;
	int y=blockIdx.y*blockDim.y + threadIdx.y;
	C[y][x]=A[y][x]+B[y][x];//subtraction
}
int main(){
	int* h_A, h_B, h_C; //declaration pointers for host memory
	int* g_A, g_B, g_C; //pointers for global memory on the device
	//host memory allocation
	h_A = (int*) malloc (H*W*sizeof(int)); 
	h_B = (int*) malloc (H*W*sizeof(int)); 
	h_C = (int*) malloc (H*W*sizeof(int)); 
	//device memory allocation
	hipMalloc((void**)&g_A, H*W*sizeof(int)); 
	hipMalloc((void**)&g_B, H*W*sizeof(int));
	hipMalloc((void**)&g_C, H*W*sizeof(int));
	for(int r=0;r<H;r++){
		for(int c=0;c<W;c++){
			h_A[r*W+c]=r*W+c;
			h_B[r*W+c]=1;
		}
	}		
	//copying h_A to g_A
	hipMemcpy( g_A, h_A, H*W*sizeof(int),hipMemcpyHostToDevice);
	//copying h_B to g_B
	hipMemcpy( g_B, h_B, H*W*sizeof(int),hipMemcpyHostToDevice);
	//define block size and grid size
	dim3 BlockSize(16,16,1);
	dim3 GridSize(W/dimBlock.x,H/dimBlock.y,1);
	//proceed the kernel function
	kernelAdd<<<GridSize,BlockSize>>>(g_A,g_B,g_C);
	//copying g_C to h_C
	hipMemcpy(h_C,g_C,h*w*sizeof(int),hipMemcpyDeviceToHost);
	//deallocate space in memory
	free(h_A);free(h_B);free(h_C);
	hipFree(g_A);hipFree(g_B);hipFree(g_C);
}