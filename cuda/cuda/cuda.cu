//cuda.cu
//simple CUDA functions
//by Wasit 20-8-2011
#include <stdio.h>
#include <hip/hip_runtime.h>
//memory allocation on device side
extern "C" void CUDA_Constructor(int** g_A,int** g_B,int size){
    hipMalloc(g_A,sizeof(int)*size);
    hipMalloc(g_B,sizeof(int)*size);
}
//copying data from host to device
extern "C" void CUDA_SetData(int* g_dist, int* h_src,int size){
    hipMemcpy(g_dist,h_src,sizeof(int)*size,hipMemcpyHostToDevice);
}
//CUDA Kernel block and thread ID are indicated by blockIdx and threadIdx, respectively
__global__ void Kernel_Add(int* g_A,int* g_B){
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    g_A[x]=g_A[x]+g_B[x];
}
//Addition function
//number of thread and block is set before call Kernel
extern "C" void CUDA_Add(int* g_A,int* g_B,int size){
    int threadnum=16;
    int blocknum=size/threadnum;
    Kernel_Add<<<threadnum,blocknum>>>(g_A,g_B);
}
//read data back to host
extern "C" void CUDA_GetData(int* h_dist, int* g_src,int size){
    hipMemcpy(h_dist,g_src,sizeof(int)*size,hipMemcpyDeviceToHost);
}
