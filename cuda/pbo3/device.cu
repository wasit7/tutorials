#include "hip/hip_runtime.h"

//#include <cutil.h>
//#include <cutil_math.h>
//#include <cutil_inline_runtime.h>
#include "device.h"
__global__ void g_createVertices(float4* positions, float time, unsigned int width, unsigned int height){
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;
    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time)
    * cosf(v * freq + time) * 0.5f;
    // Write positions
    positions[y * width + x] = make_float4(u, w, v, 1.0f);
}
extern "C" void createVertices(float4* positions, float time,unsigned int width, unsigned int height){
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    g_createVertices<<<dimGrid, dimBlock>>>(positions, time,width, height);
    //createPixels<<<dimGrid,dimBlock>>>(ptr,time,width,height);
    //cutilSafeCall( hipDeviceSynchronize() );
}
