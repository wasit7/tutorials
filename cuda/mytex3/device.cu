#include "hip/hip_runtime.h"
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "device.h"
texture<uint2, 2, hipReadModeElementType> atlasTexture;
__global__ void kernel_render(uchar4 *d_output, uint width, uint height){
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float) width;
    float v = y / (float) height;

    if ((x < width) && (y < height))
    {
        uchar4 color=make_uchar4(u*255,v*255,127,128);
        uint i = y * width + x;
        d_output[i] = color;
    }
}

extern "C"
void device_render(uchar4 *d_output, uint width, uint height)
{

    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(width / blockSize.x, height / blockSize.y);
    kernel_render<<<gridSize, blockSize>>>(d_output, width, height);

}
